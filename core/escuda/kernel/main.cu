
#include <hip/hip_runtime.h>
#include <cmath>
// #include <map>
#include <vector>
// #include <sstream>
// #include <stdexcept>
// #include <cstdlib>

__global__ void multiply_them(float *dest, float *a, float *b){
    const int i = threadIdx.x;
    dest[i] = a[i] * b[i];
}
